#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <argp.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <npp.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "gxkernel.h"

/*
 * Code to test the kernels in the gxkernel.cu.
 */

void preLaunchCheck() {
  hipError_t error;

  error = hipGetLastError();
  
  if (error != hipSuccess) {
    fprintf(stderr, "Error: Previous CUDA failure: \"%s\". Exiting\n",
	    hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

void postLaunchCheck() {
  hipError_t error;

  error = hipGetLastError();
  
  if (error != hipSuccess) {
    fprintf(stderr, "Error: Failure Launching kernel: \"%s\". Exiting\n",
	    hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

const char *argp_program_version = "benchmark_gxkernel 1.0";
static char doc[] = "benchmark_gxkernel -- testing performance of various kernels";
static char args_doc[] = "";

/* Our command line options */
static struct argp_option options[] = {
  { "loops", 'n', "NLOOPS", 0, "run each performance test NLOOPS times" },
  { "threads", 't', "NTHREADS", 0, "run with NTHREADS threads on each test" },
  { "antennas", 'a', "NANTENNAS", 0, "assume NANTENNAS antennas when required" },
  { "channels", 'c', "NCHANNELS", 0, "assume NCHANNELS frequency channels when required" },
  { "samples", 's', "NSAMPLES", 0, "assume NSAMPLES when unpacking" },
  { "bandwidth", 'b', "BANDWIDTH", 0, "the bandwidth in Hz" },
  { "verbose", 'v', 0, 0, "output more" },
  { "bits", 'B', "NBITS", 0, "number of bits assumed in the data" },
  { "complex", 'I', 0, 0, "the data input is complex sampled" },
  { 0 }
};

struct arguments {
  int nloops;
  int nthreads;
  int nantennas;
  int nchannels;
  int nsamples;
  int bandwidth;
  int verbose;
  int nbits;
  int complexdata;
};

/* The option parser */
static error_t parse_opt(int key, char *arg, struct argp_state *state) {
  struct arguments *arguments = (struct arguments *)state->input;

  switch (key) {
  case 'n':
    arguments->nloops = atoi(arg);
    break;
  case 't':
    arguments->nthreads = atoi(arg);
    break;
  case 'a':
    arguments->nantennas = atoi(arg);
    break;
  case 'c':
    arguments->nchannels = atoi(arg);
    break;
  case 's':
    arguments->nsamples = atoi(arg);
    break;
  case 'b':
    arguments->bandwidth = atoi(arg);
    break;
  case 'v':
    arguments->verbose = 1;
    break;
  case 'B':
    arguments->nbits = atoi(arg);
    break;
  case 'C':
    arguments->complexdata = 1;
    break;
  }
  return 0;
}

/* The argp parser */
static struct argp argp = { options, parse_opt, args_doc, doc };

void time_stats(float *timearray, int ntime, float *average, float *min, float *max) {
  int i = 0;
  *average = 0.0;
  for (i = 1; i < ntime; i++) {
    *average += timearray[i];
    if (i == 1) {
      *min = timearray[i];
      *max = timearray[i];
    } else {
      *min = (timearray[i] < *min) ? timearray[i] : *min;
      *max = (timearray[i] > *max) ? timearray[i] : *max;
    }
  }

  if ((ntime - 1) > 0) {
    *average /= (float)(ntime - 1);
  }
  return;
}

#define NOFRINGEROTATE

int main(int argc, char *argv[]) {
  
  /* Default argument values first. */
  struct arguments arguments;
  arguments.nloops = 100;
  arguments.nthreads = 512;
  arguments.nantennas = 6;
  arguments.nchannels = 2048;
  arguments.nsamples = 1<<23;
  arguments.bandwidth = 64e6;
  arguments.verbose = 0;
  arguments.nbits = 2;
  arguments.complexdata = 0;
  int npolarisations = 2;
  
  argp_parse(&argp, argc, argv, 0, 0, &arguments);

  // Always discard the first trial.
  arguments.nloops += 1;
  
  printf("BENCHMARK PROGRAM STARTS\n\n");

#ifndef NOUNPACK
  /*
   * This benchmarks unpacker kernels.
   */
  hipComplex **unpacked = new hipComplex*[arguments.nantennas * npolarisations];
  hipComplex *unpacked2 = new hipComplex[arguments.nantennas * npolarisations * arguments.nsamples];
  hipComplex **unpackedData;
  int8_t **packedData;
  float *dtime_unpack=NULL, *dtime_unpack2=NULL; 
  float averagetime_unpack = 0.0, mintime_unpack = 0.0, maxtime_unpack = 0.0;
  float averagetime_unpack2 = 0.0, mintime_unpack2 = 0.0, maxtime_unpack2 = 0.0;
  float implied_time;
  hipEvent_t start_test_unpack, end_test_unpack;
  hipEvent_t start_test_unpack2, end_test_unpack2;
  hiprandGenerator_t gen;
  dtime_unpack = (float *)malloc(arguments.nloops * sizeof(float));
  dtime_unpack2 = (float *)malloc(arguments.nloops * sizeof(float));
  int i, j, unpackBlocks;

  // Allocate the memory.
  int packedBytes = arguments.nsamples * 2 * npolarisations / 8;
  packedData = new int8_t*[arguments.nantennas];
  for (i = 0; i < arguments.nantennas; i++) {
    gpuErrchk(hipMalloc(&packedData[i], packedBytes));
  }

  for (i = 0; i < arguments.nantennas * npolarisations; i++) {
    gpuErrchk(hipMalloc(&unpacked[i], arguments.nsamples * sizeof(hipComplex)));
  }
  gpuErrchk(hipMalloc(&unpackedData, arguments.nantennas * npolarisations * sizeof(hipComplex*)));
  gpuErrchk(hipMemcpy(unpackedData, unpacked, arguments.nantennas * npolarisations * sizeof(hipComplex*), hipMemcpyHostToDevice));

  /*for (i = 0; i < arguments.nantennas; i++) {
    gpuErrchk(hipMalloc(&unpacked2[i], arguments.nsamples * npolarisations * sizeof(hipComplex)));
    }*/
  
  unpackBlocks = arguments.nsamples / npolarisations / arguments.nthreads;
  printf("Each test will run with %d threads, %d blocks\n", arguments.nthreads, unpackBlocks);
  printf("  nsamples = %d\n", arguments.nsamples);
  printf("  nantennas = %d\n", arguments.nantennas);
  
  hipEventCreate(&start_test_unpack);
  hipEventCreate(&end_test_unpack);
  hipEventCreate(&start_test_unpack2);
  hipEventCreate(&end_test_unpack2);
  // Generate some random data.
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  for (i = 0; i < arguments.nantennas; i++) {
    hiprandGenerateUniform(gen, (float*)packedData[i], packedBytes * (sizeof(int8_t) / sizeof(float)));
  }
  hiprandDestroyGenerator(gen);
  for (i = 0; i < arguments.nloops; i++) {
    if (arguments.verbose) {
      printf("\nLOOP %d\n", i);
    }

    // Now do the unpacking.
    preLaunchCheck();
    if (arguments.verbose) {
      printf("  RUNNING KERNEL... ");
    }
    hipEventRecord(start_test_unpack, 0);
    for (j = 0; j < arguments.nantennas; j++) {
      old_unpack2bit_2chan<<<unpackBlocks, arguments.nthreads>>>(unpackedData, packedData[j], j);
    }
    hipEventRecord(end_test_unpack, 0);
    hipEventSynchronize(end_test_unpack);
    hipEventElapsedTime(&(dtime_unpack[i]), start_test_unpack, end_test_unpack);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", dtime_unpack[i]);
    }
    postLaunchCheck();

    preLaunchCheck();
    if (arguments.verbose) {
      printf("  RUNNING KERNEL 2... ");
    }
    hipEventRecord(start_test_unpack2, 0);
    for (j = 0; j < arguments.nantennas; j++) {
      unpack2bit_2chan<<<unpackBlocks, arguments.nthreads>>>(&unpacked2[2*j*arguments.nsamples], packedData[j]);
    }
    hipEventRecord(end_test_unpack2, 0);
    hipEventSynchronize(end_test_unpack2);
    hipEventElapsedTime(&(dtime_unpack2[i]), start_test_unpack2, end_test_unpack2);
    if (arguments.verbose) {
      printf("  done in %8.3f ms.\n", dtime_unpack2[i]);
    }
    postLaunchCheck();
  }
  (void)time_stats(dtime_unpack, arguments.nloops, &averagetime_unpack,
		   &mintime_unpack, &maxtime_unpack);
  (void)time_stats(dtime_unpack2, arguments.nloops, &averagetime_unpack2,
		   &mintime_unpack2, &maxtime_unpack2);
  implied_time = (float)arguments.nsamples;
  if (arguments.complexdata) {
    // Bandwidth is the same as the sampling rate.
    implied_time /= (float)arguments.bandwidth;
    // But the data is twice as big.
    implied_time /= 2;
  } else {
    implied_time /= 2 * (float)arguments.bandwidth;
  }
  printf("\n==== ROUTINE: old_unpack2bit_2chan ====\n");
  printf("Iterations | Average time |  Min time   |  Max time   | Data time  | Speed up  |\n");
  printf("%5d      | %8.3f ms  | %8.3f ms | %8.3f ms | %8.3f s | %8.3f  |\n", (arguments.nloops - 1),
	 averagetime_unpack, mintime_unpack, maxtime_unpack, implied_time,
	 ((implied_time * 1e3) / averagetime_unpack));
  printf("\n==== ROUTINE: unpack2bit_2chan ====\n");
  printf("Iterations | Average time |  Min time   |  Max time   | Data time  | Speed up  |\n");
  printf("%5d      | %8.3f ms  | %8.3f ms | %8.3f ms | %8.3f s | %8.3f  |\n", (arguments.nloops - 1),
	 averagetime_unpack2, mintime_unpack2, maxtime_unpack2, implied_time,
	 ((implied_time * 1e3) / averagetime_unpack2));
  
  
  // Clean up.
  hipEventDestroy(start_test_unpack);
  hipEventDestroy(end_test_unpack);
  hipEventDestroy(start_test_unpack2);
  hipEventDestroy(end_test_unpack2);

#endif
  
  
#ifndef NOFRINGEROTATE
  /*
   * This benchmarks the performance of the fringe rotator kernel.
   */
  hipComplex *unpackedFR = new hipComplex*[arguments.nantennas * npolarisations * arguments.nsamples];
  int i, j, k;
  float *dtime_addcomplex=NULL, averagetime_addcomplex = 0.0;
  float mintime_addcomplex = 0.0, maxtime_addcomplex = 0.0;
  hipEvent_t start_test_addcomplex, end_test_addcomplex;
  dtime_addcomplex = (float *)malloc(arguments.nloops * sizeof(float));

  // Prepare the large arrays.
  for (i = 0; i < arguments.nantennas * npolarisations; i++) {
    gpuErrchk(hipMalloc(&unpacked[i], arguments.nchannels * sizeof(hipComplex)));
  }
  for (i = 0; i < arguments.nloops; i++) {
    // Set the complex number values.
    for (j = 0; j < arguments.nantennas * npolarisations; j++) {
      for (k = 0; k < arguments.nchannels; k++) {
	unpacked[j][k] = make_hipComplex(4.0 * ((float)rand() / (float)RAND_MAX),
					4.0 * ((float)rand() / (float)RAND_MAX));
      }
    }
    
    preLaunchCheck();
    hipEventRecord(start_test_addcomplex, 0);
    cuCaddIf<<<1, arguments.nthreads>>(&a, b);
    hipEventRecord(stop_test_addcomplex, 0);
    hipEventSynchronize(stop_test_addcomplex);
    hipEventElapsedTime(&(dtime_addcomplex[i]), start_test_addcomplex, stop_test_addcomplex);
    postLaunchCheck();
  }
  // Do some statistics.
  (void)time_stats(dtime_addcomplex, arguments.nloops, &averagetime_addcomplex,
		   &mintime_addcomplex, &maxtime_addcomplex);
  printf("\n==== ROUTINE: cuCaddIf ====\n");
  printf("Iterations | Average time | Min time | Max time |\n");
  printf("%5d     | %8.3f ms | %8.3f ms | %8.3f ms |\n", arguments.nloops,
	 averagetime_addcomplex, mintime.addcomplex, maxtime_addcomplex);
  hipEventDestroy(start_test_addcomplex);
  hipEventDestroy(end_test_addcomplex);
#endif
  
}


