#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <string>
#include <vector>
#include <cstdlib>
#include <cstring>
#include <stdint.h>
#include <complex>
#include <argp.h>

#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#include "common.h"

#define NTHREADS 256

using std::string;
using std::cout;
using std::cerr;
using std::endl;
using std::vector;

const char *argp_program_version = "testgpukernel 1.0";
static char doc[] = "testgpukernel -- testing operation of the GPU correlator code";
static char args_doc[] = "configuration_file";

#define BUFSIZE 256

/* Our command line options */
static struct argp_option options[] = {
  { "loops", 'n', "NLOOPS", 0, "run the code N times in a loop" },
  { "binary", 'b', 0, 0, "output binary instead of default text" },
  { 0 }
};

struct arguments {
  int output_binary;
  int nloops;
  char configfile[BUFSIZE];
};

/* The option parser */
static error_t parse_opt(int key, char *arg, struct argp_state *state) {
  struct arguments *arguments = (struct arguments *)state->input;

  switch (key) {
  case 'b':
    arguments->output_binary = 1;
    break;
  case 'n':
    arguments->nloops = atoi(arg);
    break;
  case ARGP_KEY_END:
    if (strlen(arguments->configfile) == 0) {
      argp_usage(state);
      exit(0);
    }
    break;
  default:
    // Assume this is the config file.
    if (arg != NULL) {
       if (strlen(arg) > 0) {
       	  strncpy(arguments->configfile, arg, BUFSIZE);
       }
    }
  }
  return 0;
}

/* The argp parser */
static struct argp argp = { options, parse_opt, args_doc, doc };


#include "gxkernel.h"

void allocDataGPU(int8_t ***packedData, hipComplex **unpackedData,
		  hipComplex **channelisedData, hipComplex **baselineData, 
		  float **rotationPhaseInfo, float **fractionalSampleDelays, int **sampleShifts, 
                  double **gpuDelays, int numantenna, int subintsamples, int nbit, 
                  int nPol, bool iscomplex, int nchan, int numffts, int parallelAccum)
{
  unsigned long long GPUalloc = 0;

  int packedBytes = (subintsamples+nchan*2)*nbit*nPol/8; // Allow a little extra for delay shift
  *packedData = new int8_t*[numantenna];
  
  for (int i=0; i<numantenna; i++) {
    gpuErrchk(hipMalloc(&(*packedData)[i], packedBytes));
    GPUalloc += packedBytes;
  }

  // Unpacked data
  gpuErrchk(hipMalloc(unpackedData, numantenna*nPol*subintsamples*sizeof(hipComplex)));
  GPUalloc += numantenna*nPol*subintsamples*sizeof(hipComplex);
  
  // FFT output
  gpuErrchk(hipMalloc(channelisedData, numantenna*nPol*subintsamples*sizeof(hipComplex)));
  GPUalloc += numantenna*nPol*subintsamples*sizeof(hipComplex);

  // Baseline visibilities
  int nbaseline = numantenna*(numantenna-1)/2;
  if (!iscomplex) subintsamples /= 2;
  cout << "Alloc " << nchan*parallelAccum << " complex output values per baseline" << endl;
  gpuErrchk(hipMalloc(baselineData, nbaseline*4*nchan*parallelAccum*sizeof(hipComplex)));
  GPUalloc += nbaseline*4*nchan*parallelAccum*sizeof(hipComplex);

  // Fringe rotation vector (will contain starting phase and phase increment for every FFT of every antenna)
  gpuErrchk(hipMalloc(rotationPhaseInfo, numantenna*numffts*2*sizeof(float)));
  GPUalloc += numantenna*numffts*2*sizeof(float);

  // Fractional sample delay vector (will contain midpoint fractional sample delay [in units of radians per channel!] 
  // for every FFT of every antenna)
  gpuErrchk(hipMalloc(fractionalSampleDelays, numantenna*numffts*sizeof(float)));
  GPUalloc += numantenna*numffts*sizeof(float);

  // Sample shifts vector (will contain the integer sample shift relative to nominal FFT start for every FFT of every antenna)
  gpuErrchk(hipMalloc(sampleShifts, numantenna*numffts*sizeof(int)));
  GPUalloc += numantenna*numffts*sizeof(int);

  // Delay information vectors
  gpuErrchk(hipMalloc(gpuDelays, numantenna*4*sizeof(double)));
  GPUalloc += numantenna*4*sizeof(double);
  
  cout << "Allocated " << GPUalloc/1e6 << " Mb on GPU" << endl;
}

inline float carg(const hipComplex& z) {return atan2(hipCimagf(z), hipCrealf(z));} // polar angle

void saveVisibilities(const char *outfile, hipComplex *baselines, int nbaseline, int nchan, int stride, double bandwidth) {
  hipComplex **vis;
  std::ofstream fvis(outfile);

  // Copy final visibilities back to CPU
  vis = new hipComplex*[nbaseline*4];
  for (int i=0; i<nbaseline*4; i++) {
    vis[i] = new hipComplex[nchan];
    gpuErrchk(hipMemcpy(vis[i], &baselines[i*stride], nchan*sizeof(hipComplex), hipMemcpyDeviceToHost));
  }
  
  for (int c=0; c<nchan; c++) {
    fvis << std::setw(5) << c << " " << std::setw(11) << std::fixed << std::setprecision(6) << (c+0.5)/nchan*bandwidth/1e6;
    fvis  << std::setprecision(5);
    for (int i=0; i<nbaseline*4; i++) {
      fvis << " " << std::setw(11) << vis[i][c].x << " " << std::setw(11) << vis[i][c].y;
      fvis << " " << std::setw(11) << hipCabsf(vis[i][c]) << " " << std::setw(11) << carg(vis[i][c]);
    }
    fvis << std::endl;
  }
  fvis.close();
  
  for (int i=0;i<nbaseline*4;i++) {
    delete [] vis[i];
  }
  delete [] vis;
}

int main(int argc, char *argv[])
{
  // variables for the test
  char *configfile;
  int subintbytes, status, cfactor;
  int nPol;
  uint8_t ** inputdata;
  double ** delays; /**< delay polynomial for each antenna.  delay is in seconds, time is in units of FFT duration */
  double * antfileoffsets; /**< offset from each the nominal start time of the integration for each antenna data file.  
                                In units of seconds. */
  int numchannels, numantennas, nbaseline, numffts, nbit;
  double lo, bandwidth, sampletime, subinttime;
  bool iscomplex;
  vector<string> antennas, antFiles;
  vector<std::ifstream *> antStream;

  int8_t **packedData;
  float *rotationPhaseInfo;
  float *fractionalSampleDelays;
  int *sampleShifts;
  double *gpuDelays;
  hipComplex *unpackedData, *channelisedData, *baselineData;
  hipfftHandle plan;
  hipEvent_t start_exec, stop_exec;
  
  // Read in the command line arguments.
  struct arguments arguments;
  arguments.nloops = 1;
  arguments.output_binary = 0;
  arguments.configfile[0] = 0;
  argp_parse(&argp, argc, argv, 0, 0, &arguments);

  if (strlen(arguments.configfile) > 0) {
    configfile = arguments.configfile;
  }
  printf("reading configuration file %s\n", arguments.configfile);
  printf("running %d loops\n", arguments.nloops);
  printf("will output %s data\n", (arguments.output_binary == 0) ? "text" : "binary");

  hipEventCreate(&start_exec);
  hipEventCreate(&stop_exec);
  
  init_2bitLevels();

  // load up the test input data and delays from the configfile
  parseConfig(configfile, nbit, nPol, iscomplex, numchannels, numantennas, lo, bandwidth, numffts, antennas, antFiles, &delays, &antfileoffsets);

  nbaseline = numantennas*(numantennas-1)/2;
  if (iscomplex) {
    cfactor = 1;
  } else{
    cfactor = 2; // If real data FFT size twice size of number of frequecy channels
  }

  int fftchannels = numchannels*cfactor;
  int subintsamples = numffts*fftchannels;  // Number of time samples - need to factor # channels (pols) also
  cout << "Subintsamples= " << subintsamples << endl;

  sampletime = 1.0/bandwidth;
  if (!iscomplex) sampletime /= 2.0; 
  subinttime = subintsamples*sampletime;
  cout << "Subint = " << subinttime*1000 << " msec" << endl;

  // Setup threads and blocks for the various kernels
  // Unpack
  int unpackThreads = NTHREADS;
  int unpackBlocks;
  if (nbit==2 && !iscomplex) {
    unpackBlocks = subintsamples/2/unpackThreads; // 2 time samples/byte
  } else if (nbit==8 && iscomplex) {
    unpackBlocks = subintsamples*nPol; // Each pol separately
  } else {
    cerr << "Error: Unsupported number if bits/complex (" << nbit << "/" << iscomplex << ")" << endl;
    exit(1);
  }
  if (unpackThreads*unpackBlocks*nPol!=subintsamples) {
    cerr << "Error: <<" << unpackBlocks << "," << unpackThreads << ">> inconsistent with " << subintsamples << " samples for unpack kernel" << endl;
  }

  // Fringe Rotate
  int fringeThreads, blockchan;
  if (fftchannels<=NTHREADS) {
    fringeThreads = fftchannels;
    blockchan = 1;
  } else {
    fringeThreads = NTHREADS;
    blockchan = fftchannels/NTHREADS;
    if (fftchannels%NTHREADS) {
      cerr << "Error: NTHREADS not divisible into fftchannels" << endl;
      exit(1);
    }
  }
  dim3 fringeBlocks = dim3(blockchan, numffts, numantennas);

  // Fractional Delay
  int fracDelayThreads;
  if (numchannels<=NTHREADS) {
    fracDelayThreads = numchannels;
    blockchan = 1;
  } else {
    fracDelayThreads = NTHREADS;
    blockchan = numchannels/NTHREADS;
    if (numchannels%NTHREADS) {
      cerr << "Error: NTHREADS not divisible into fftchannels" << endl;
      exit(1);
    }
  }
  dim3 fracDelayBlocks = dim3(blockchan, numffts, numantennas);

  // CrossCorr
  int targetThreads = 50e4;  // This seems a *lot*
  int corrThreads;
  if (numchannels<=512) {
    corrThreads = numchannels;
    blockchan = 1;
  } else {
    corrThreads = 512;
    blockchan = numchannels/512;
  }
  int parallelAccum = (int)ceil(targetThreads/numchannels+1); // I suspect this has failure modes
  cout << "Initial parallelAccum=" << parallelAccum << endl;
  while (parallelAccum && numffts % parallelAccum) parallelAccum--;
  if (parallelAccum==0) {
    cerr << "Error: Could not determine block size for Cross Correlation" << endl;
    exit(1);
  }
  int nchunk = numffts / parallelAccum;
  dim3 corrBlocks = dim3(blockchan, parallelAccum);
  cout << "Corr Threads:  " << corrThreads << " " << blockchan << ":" << parallelAccum << "/" << nchunk << endl;

  // Final Cross Corr accumulation
  dim3 accumBlocks = dim3(blockchan, 4, nbaseline);

  
  cout << "Allocate Memory" << endl;
  // Allocate space in the buffers for the data and the delays
  allocDataHost(&inputdata, numantennas, numchannels, numffts, nbit, nPol, iscomplex, subintbytes);

  // Allocate space on the GPU
  allocDataGPU(&packedData, &unpackedData, &channelisedData,
	       &baselineData, &rotationPhaseInfo, &fractionalSampleDelays, &sampleShifts,
               &gpuDelays, numantennas, subintsamples,
	       nbit, nPol, iscomplex, numchannels, numffts, parallelAccum);

  for (int i=0; i<numantennas; i++) {
    antStream.push_back(new std::ifstream(antFiles[i].c_str(), std::ios::binary));
  }

  // Configure CUFFT
  if (hipfftPlan1d(&plan, fftchannels, HIPFFT_C2C, nPol*numantennas*numffts) != HIPFFT_SUCCESS) {
    cout << "CUFFT error: Plan creation failed" << endl;
    return(0);
  }
  
  status = readdata(subintbytes, antStream, inputdata);
  if (status) exit(1);

  // Copy data to GPU
  cout << "Copy data to GPU" << endl;
  for (int i=0; i<numantennas; i++) {
    gpuErrchk(hipMemcpy(packedData[i], inputdata[i], subintbytes, hipMemcpyHostToDevice)); 
  }

  // Copy delays to GPU
  cout << "Copy delays to GPU" << endl;
  for (int i=0; i<numantennas; i++) {
    gpuErrchk(hipMemcpy(&(gpuDelays[i*4]), delays[i], 3*sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(&(gpuDelays[i*4+3]), &(antfileoffsets[i]), sizeof(double), hipMemcpyHostToDevice));
  }

  // Check that the number of FFTs is a valid number
  if (numffts%8)
  {
    cerr << "Error: numffts must be divisible by 8" << endl;
    exit(1);
  }
  // Set the number of blocks for fringe rotation (and fractional sample delay?)
  dim3 FringeSetblocks = dim3(8, numantennas);

  // Record the start time
  hipEventRecord(start_exec, 0);
  for (int l=0; l<arguments.nloops; l++)
  {
    // Use the delays to calculate fringe rotation phases and fractional sample delays for each FFT //
    calculateDelaysAndPhases<<<FringeSetblocks, numffts/8>>>(gpuDelays, lo, sampletime, fftchannels, numchannels, rotationPhaseInfo, 
                                                             sampleShifts, fractionalSampleDelays);
    CudaCheckError();

    // Unpack the data
    //cout << "Unpack data" << endl;
    for (int i=0; i<numantennas; i++) {
      if (nbit==2 && !iscomplex) {
	unpack2bit_2chan_fast<<<unpackBlocks,unpackThreads>>>(&unpackedData[2*i*subintsamples], packedData[i], &(sampleShifts[numffts*i]));
      } else if (nbit==8 && iscomplex) {
	unpack8bitcomplex_2chan<<<unpackBlocks,unpackThreads>>>(&unpackedData[2*i*subintsamples], packedData[i]);
      }
      CudaCheckError();
    }

    /*// Fringe Rotate //
    //cout << "Fringe Rotate" << endl;
    setFringeRotation<<<FringeSetblocks, numffts/8>>>(rotationPhaseInfo);
    CudaCheckError();*/

    FringeRotate<<<fringeBlocks,fringeThreads>>>(unpackedData, rotationPhaseInfo);
    CudaCheckError();
  
    // FFT
    //cout << "FFT data" << endl;
    if (hipfftExecC2C(plan, unpackedData, channelisedData, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
      cout << "CUFFT error: ExecC2C Forward failed" << endl;
      return(0);
    }

    // Fractional Delay Correction
    //FracSampleCorrection<<<fracDelayBlocks,fracDelayThreads>>>(channelisedData, fractionalDelayValues, numchannels, fftchannels, numffts, subintsamples);
    //CudaCheckError();
    
    // Cross correlate
    gpuErrchk(hipMemset(baselineData, 0, nbaseline*4*numchannels*parallelAccum*sizeof(hipComplex)));

#if 0
    cout << "Cross correlate" << endl;
    CrossCorr<<<corrBlocks,corrThreads>>>(channelisedData, baselineData, numantennas, nchunk);
    CudaCheckError();
    // cout << "Finalise" << endl;
    finaliseAccum<<<accumBlocks,corrThreads>>>(baselineData, parallelAccum, nchunk);
    CudaCheckError();
#elif 0
    int ccblock_width = 128;
    dim3 ccblock(1+(numchannels-1)/ccblock_width, numantennas-1, numantennas-1);
    CrossCorrAccumHoriz<<<ccblock, ccblock_width>>>(baselineData, channelisedData, numantennas, numffts, numchannels, fftchannels);
#else
    int ccblock_width = 128;
    int nantxp = numantennas*2;
    dim3 ccblock(1+(numchannels-1)/ccblock_width, nantxp-1, nantxp-1);
    CCAH2<<<ccblock, ccblock_width>>>(baselineData, channelisedData, numantennas, numffts, numchannels, fftchannels);
#endif
  }
  
  float dtime;
  hipEventRecord(stop_exec, 0);
  hipEventSynchronize(stop_exec);
  hipEventElapsedTime(&dtime, start_exec, stop_exec);

  cout << "Total execution time for " << arguments.nloops << " loops =  " <<  dtime << " ms" << endl;

#if 0
  saveVisibilities("vis.out", baselineData, nbaseline, numchannels, parallelAccum*numchannels, bandwidth);
#else
  saveVisibilities("vis.out", baselineData, nbaseline, numchannels, numchannels, bandwidth);
#endif

  hipDeviceSynchronize();
  hipDeviceReset();

  // Calculate the elapsed time

  // Free memory
  //  for (i=0; i<numantennas; i++)
  //{
  //  delete(inputdata[i]);
  //  delete(delays[i]);
  //}
  //delete(inputdata);
  //delete(delays);
}
